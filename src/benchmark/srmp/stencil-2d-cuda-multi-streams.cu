#include "hip/hip_runtime.h"
#include "stencil-2d-cuda-util.cuh"
#include "stencil-2d-util.h"

#include "../../cuda-util.h"


template <typename tpe>
__global__ void stencil2d(const tpe *const __restrict__ u, tpe *__restrict__ uNew, const size_t nx, const size_t ny) {
    const size_t i0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
    const size_t i1 = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (i0 < nx - 1 && i1 < ny - 1) {
        uNew[i0 + i1 * nx] = 0.25 * u[i0 + i1 * nx + 1] + 0.25 * u[i0 + i1 * nx - 1] + 0.25 * u[i0 + nx * (i1 + 1)] + 0.25 * u[i0 + nx * (i1 - 1)];
    }
}

template <typename tpe>
inline void performIteration(tpe **&h_d_u, tpe **&h_d_uNew, const size_t nx, const size_t ny,
                             int patch_nx, int patch_ny,
                             hipStream_t *streams) {

    dim3 blockSize(16, 16);
    dim3 numBlocks(ceilingDivide(nx - 2, blockSize.x), ceilingDivide(ny - 2, blockSize.y));
    int blockSize1D = 32;

    for (auto py = 0; py < patch_ny; ++py) {
        for (auto px = 0; px < patch_nx; ++px) {
            auto patch = py * patch_nx + px;
            if (px > 0)
                exchangeWest<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D, 0, streams[5 * patch + 0]>>>(h_d_u[patch], h_d_u[py * patch_nx + px - 1], nx, ny);
            else
                applyBCWest<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D, 0, streams[5 * patch + 0]>>>(h_d_u[patch], nx, ny);

            if (px < patch_nx - 1)
                exchangeEast<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D, 0, streams[5 * patch + 1]>>>(h_d_u[patch], h_d_u[py * patch_nx + px + 1], nx, ny);
            else
                applyBCEast<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D, 0, streams[5 * patch + 1]>>>(h_d_u[patch], nx, ny);

            if (py > 0)
                exchangeSouth<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D, 0, streams[5 * patch + 2]>>>(h_d_u[patch], h_d_u[(py - 1) * patch_nx + px], nx, ny);
            else
                applyBCSouth<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D, 0, streams[5 * patch + 2]>>>(h_d_u[patch], nx, ny);

            if (py < patch_ny - 1)
                exchangeNorth<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D, 0, streams[5 * patch + 3]>>>(h_d_u[patch], h_d_u[(py + 1) * patch_nx + px], nx, ny);
            else
                applyBCNorth<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D, 0, streams[5 * patch + 3]>>>(h_d_u[patch], nx, ny);
        }
    }

    checkCudaError(hipDeviceSynchronize(), true);

    for (auto py = 0; py < patch_ny; ++py) {
        for (auto px = 0; px < patch_nx; ++px) {
            auto patch = px + py * patch_nx;
            stencil2d<<<numBlocks, blockSize, 0, streams[5 * patch + 4]>>>(h_d_u[patch], h_d_uNew[patch], nx, ny);
        }
    }

    checkCudaError(hipDeviceSynchronize(), true);

    std::swap(h_d_u, h_d_uNew);
}

template <typename tpe>
inline int realMain(int argc, char *argv[]) {
    char *tpeName;
    size_t nx, ny, nItWarmUp, nIt;
    unsigned int patch_nx, patch_ny;
    parseCLA_2d(argc, argv, tpeName, nx, ny, nItWarmUp, nIt, patch_nx, patch_ny);

    auto numPatches = patch_nx * patch_ny;

    tpe **u;
    checkCudaError(hipHostMalloc((void **)&u, sizeof(tpe *) * numPatches));
    tpe **uNew;
    checkCudaError(hipHostMalloc((void **)&uNew, sizeof(tpe *) * numPatches));
    for (auto i = 0; i < numPatches; ++i) {
        checkCudaError(hipHostMalloc((void **)&u[i], sizeof(tpe) * nx * ny));
        checkCudaError(hipHostMalloc((void **)&uNew[i], sizeof(tpe) * nx * ny));
    }

    tpe **h_d_u;
    tpe **h_d_uNew;
    checkCudaError(hipHostMalloc((void **)&h_d_u, sizeof(tpe *) * numPatches));
    checkCudaError(hipHostMalloc((void **)&h_d_uNew, sizeof(tpe *) * numPatches));
    for (auto i = 0; i < numPatches; ++i) {
        checkCudaError(hipMalloc((void **)&h_d_u[i], sizeof(tpe) * nx * ny));
        checkCudaError(hipMalloc((void **)&h_d_uNew[i], sizeof(tpe) * nx * ny));
    }

    tpe **d_d_u;
    tpe **d_d_uNew;
    checkCudaError(hipMalloc((void **)&d_d_u, sizeof(tpe *) * numPatches));
    checkCudaError(hipMalloc((void **)&d_d_uNew, sizeof(tpe *) * numPatches));
    checkCudaError(hipMemcpy(d_d_u, h_d_u, sizeof(tpe *) * numPatches, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_d_uNew, h_d_uNew, sizeof(tpe *) * numPatches, hipMemcpyHostToDevice));

    hipStream_t *streams;
    checkCudaError(hipHostMalloc((void **)&streams, sizeof(hipStream_t) * numPatches));
    for (auto i = 0; i < 5 * numPatches; ++i)
        checkCudaError(hipStreamCreate(&streams[i]));

    // init
    initStencil2D(u, uNew, nx, ny, patch_nx, patch_ny);

    for (auto i = 0; i < numPatches; ++i) {
        checkCudaError(hipMemcpy(h_d_u[i], u[i], sizeof(tpe) * nx * ny, hipMemcpyHostToDevice));
        checkCudaError(hipMemcpy(h_d_uNew[i], uNew[i], sizeof(tpe) * nx * ny, hipMemcpyHostToDevice));
    }

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i)
        performIteration(h_d_u, h_d_uNew, nx, ny, patch_nx, patch_ny, streams);
    checkCudaError(hipDeviceSynchronize(), true);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i)
        performIteration(h_d_u, h_d_uNew, nx, ny, patch_nx, patch_ny, streams);
    checkCudaError(hipDeviceSynchronize(), true);

    auto end = std::chrono::steady_clock::now();

    printStats<tpe>(end - start, nIt, numPatches * nx * ny, tpeName, sizeof(tpe) + sizeof(tpe), 7);

    for (auto i = 0; i < numPatches; ++i) {
        checkCudaError(hipMemcpy(u[i], h_d_u[i], sizeof(tpe) * nx * ny, hipMemcpyDeviceToHost));
        checkCudaError(hipMemcpy(uNew[i], h_d_uNew[i], sizeof(tpe) * nx * ny, hipMemcpyDeviceToHost));
    }

    // check solution
    tpe res = checkSolutionStencil2D(u, uNew, nx, ny, nIt + nItWarmUp, patch_nx, patch_ny);
    res = sqrt(res);
    std::cout << "  Final residual is " << res << std::endl;

    for (auto i = 0; i < 5 * numPatches; ++i)
        checkCudaError(hipStreamDestroy(streams[i]));
    checkCudaError(hipHostFree(streams));

    checkCudaError(hipFree(d_d_u));
    checkCudaError(hipFree(d_d_uNew));

    for (auto i = 0; i < numPatches; ++i) {
        checkCudaError(hipFree(h_d_u[i]));
        checkCudaError(hipFree(h_d_uNew[i]));
    }

    checkCudaError(hipHostFree(h_d_u));
    checkCudaError(hipHostFree(h_d_uNew));

    for (auto i = 0; i < numPatches; ++i) {
        checkCudaError(hipHostFree(u[i]));
        checkCudaError(hipHostFree(uNew[i]));
    }

    checkCudaError(hipHostFree(u));
    checkCudaError(hipHostFree(uNew));

    return 0;
}


int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cout << "Missing type specification" << std::endl;
        return -1;
    }

    std::string tpeName(argv[1]);

    if ("float" == tpeName)
        return realMain<float>(argc, argv);
    if ("double" == tpeName)
        return realMain<double>(argc, argv);

    std::cout << "Invalid type specification (" << argv[1] << "); supported types are" << std::endl;
    std::cout << "  int, long, float, double" << std::endl;
    return -1;
}
