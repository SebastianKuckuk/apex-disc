#include "hip/hip_runtime.h"
#include "stencil-2d-cuda-util.cuh"
#include "stencil-2d-util.h"

#include <mpi.h>

#include "../../cuda-util.h"
#include "../../nccl-util.h"


template <typename tpe>
__global__ void stencil2d(const tpe *const __restrict__ u, tpe *__restrict__ uNew, const size_t nx, const size_t ny) {
    const size_t i0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
    const size_t i1 = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (i0 < nx - 1 && i1 < ny - 1) {
        uNew[i0 + i1 * nx] = 0.25 * u[i0 + i1 * nx + 1] + 0.25 * u[i0 + i1 * nx - 1] + 0.25 * u[i0 + nx * (i1 + 1)] + 0.25 * u[i0 + nx * (i1 - 1)];
    }
}

template <typename tpe>
inline void performIteration(Patch<tpe> *patches, unsigned int numPatches, const size_t nx, const size_t ny,
                             int mpi_rank, int mpi_x, int mpi_y, int mpi_nx, int mpi_ny, ncclDataType_t nccl_tpe,
                             ncclComm_t *nccl_comms) {

    dim3 blockSize(16, 16);
    dim3 numBlocks(ceilingDivide(nx - 2, blockSize.x), ceilingDivide(ny - 2, blockSize.y));
    int blockSize1D = 32;

    for (auto p = 0; p < numPatches; ++p) {
        auto& patch = patches[p];

        if (BOUNDARY == patch.neighborType[0])
            applyBCWest<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(patch.d_u, nx, ny);
        else if (LOCAL == patch.neighborType[0])
            exchangeWest<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(patch.d_u, patches[patch.neighborPatchIdx[0]].d_u, nx, ny);
        else
            packBufferVertical<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(patch.d_u, patch.h_d_bufSend[0], nx + 1, nx, ny);

        if (BOUNDARY == patch.neighborType[1])            
            applyBCEast<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(patch.d_u, nx, ny);
        else if (LOCAL == patch.neighborType[1])
            exchangeEast<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(patch.d_u, patches[patch.neighborPatchIdx[1]].d_u, nx, ny);
        else
            packBufferVertical<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(patch.d_u, patch.h_d_bufSend[1], nx + nx - 2, nx, ny);

        if (BOUNDARY == patch.neighborType[2])
            applyBCSouth<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(patch.d_u, nx, ny);
        else if (LOCAL == patch.neighborType[2])
            exchangeSouth<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(patch.d_u, patches[patch.neighborPatchIdx[2]].d_u, nx, ny);
        else
            packBufferHorizontal<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(patch.d_u, patch.h_d_bufSend[2], nx + 1, nx, ny);

        if (BOUNDARY == patch.neighborType[3])
            applyBCNorth<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(patch.d_u, nx, ny);
        else if (LOCAL == patch.neighborType[3])
            exchangeNorth<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(patch.d_u, patches[patch.neighborPatchIdx[3]].d_u, nx, ny);
        else
            packBufferHorizontal<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(patch.d_u, patch.h_d_bufSend[3], (ny - 2) * nx + 1, nx, ny);
    }

    // no tags in nccl - use different communicators to emulate tags
    checkNcclError(ncclGroupStart());
    {
        for (auto p = 0; p < numPatches; ++p) {
            auto& patch = patches[p];

            for (auto i = 0; i < 4; ++i) {
                if (REMOTE == patch.neighborType[i]) {
                    checkNcclError(ncclSend(patch.h_d_bufSend[i], i < 2 ? ny - 2 : nx - 2, nccl_tpe,
                                   patch.neighborMpiRank[i], nccl_comms[p], 0));
                    checkNcclError(ncclRecv(patch.h_d_bufRecv[i], i < 2 ? ny - 2 : nx - 2, nccl_tpe,
                                   patch.neighborMpiRank[i], nccl_comms[patch.neighborPatchIdx[i]], 0));
                }
            }
        }
    }    
    checkNcclError(ncclGroupEnd());

    for (auto p = 0; p < numPatches; ++p) {
        auto& patch = patches[p];

        if (REMOTE == patch.neighborType[0])
            unpackBufferVertical<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(patch.h_d_bufRecv[0], patch.d_u, nx, nx, ny);
        if (REMOTE == patch.neighborType[1])
            unpackBufferVertical<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(patch.h_d_bufRecv[1], patch.d_u, nx + nx - 1, nx, ny);
        if (REMOTE == patch.neighborType[2])
            unpackBufferHorizontal<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(patch.h_d_bufRecv[2], patch.d_u, 1, nx, ny);
        if (REMOTE == patch.neighborType[3])
            unpackBufferHorizontal<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(patch.h_d_bufRecv[3], patch.d_u, (ny - 1) * nx + 1, nx, ny);
    }

    for (auto p = 0; p < numPatches; ++p) {
        auto& patch = patches[p];
        stencil2d<<<numBlocks, blockSize>>>(patch.d_u, patch.d_uNew, nx, ny);
    }

    for (auto p = 0; p < numPatches; ++p) {
        auto& patch = patches[p];
        std::swap(patch.d_u, patch.d_uNew);
    }
}

template <typename tpe>
inline int realMain(int argc, char *argv[], MPI_Datatype MPI_TPE, ncclDataType_t nccl_tpe) {
    MPI_Init(&argc, &argv);

    char *tpeName;
    size_t nx, ny, nItWarmUp, nIt;
    unsigned int mpi_nx, mpi_ny, patch_nx, patch_ny;
    parseCLA_2d(argc, argv, tpeName, nx, ny, nItWarmUp, nIt, mpi_nx, mpi_ny, patch_nx, patch_ny);

    int mpi_rank, mpi_size;
    MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);

    if (mpi_size != mpi_nx * mpi_ny) {
        std::cerr << "Number of MPI processes must be " << mpi_nx * mpi_ny << std::endl;
        MPI_Finalize();
        return -1;
    }

    int mpi_x = mpi_rank % mpi_nx;
    int mpi_y = mpi_rank / mpi_nx;

    int numDevices;
    checkCudaError(hipGetDeviceCount(&numDevices));
    int device = mpi_rank % numDevices;
    checkCudaError(hipSetDevice(device));

    auto numPatches = patch_nx * patch_ny;
    Patch<tpe> *patches;
    checkCudaError(hipHostMalloc((void **)&patches, sizeof(Patch<tpe>) * numPatches));

    for (auto py = 0; py < patch_ny; ++py) {
        for (auto px = 0; px < patch_nx; ++px) {
            auto p = py * patch_nx + px;
            auto& patch = patches[p];

            checkCudaError(hipHostMalloc((void **)&patch.u, sizeof(tpe) * nx * ny));
            checkCudaError(hipHostMalloc((void **)&patch.uNew, sizeof(tpe) * nx * ny));

            checkCudaError(hipMalloc((void **)&patch.d_u, sizeof(tpe) * nx * ny));
            checkCudaError(hipMalloc((void **)&patch.d_uNew, sizeof(tpe) * nx * ny));

            if (0 == px && 0 == mpi_x) {
                patch.neighborType[0] = BOUNDARY;
                patch.neighborPatchIdx[0] = -1;
                patch.neighborMpiRank[0] = -1;
            } else if (0 == px) {
                patch.neighborType[0] = REMOTE;
                patch.neighborPatchIdx[0] = py * patch_nx + (patch_nx - 1);
                patch.neighborMpiRank[0] = mpi_rank - 1;
            } else {
                patch.neighborType[0] = LOCAL;
                patch.neighborPatchIdx[0] = p - 1;
                patch.neighborMpiRank[0] = mpi_rank;
            }

            if (patch_nx - 1 == px && mpi_nx - 1 == mpi_x) {
                patch.neighborType[1] = BOUNDARY;
                patch.neighborPatchIdx[1] = -1;
                patch.neighborMpiRank[1] = -1;
            } else if (patch_nx - 1 == px) {
                patch.neighborType[1] = REMOTE;
                patch.neighborPatchIdx[1] = py * patch_nx;
                patch.neighborMpiRank[1] = mpi_rank + 1;
            } else {
                patch.neighborType[1] = LOCAL;
                patch.neighborPatchIdx[1] = p + 1;
                patch.neighborMpiRank[1] = mpi_rank;
            }

            if (0 == py && 0 == mpi_y) {
                patch.neighborType[2] = BOUNDARY;
                patch.neighborPatchIdx[2] = -1;
                patch.neighborMpiRank[2] = -1;
            } else if (0 == py) {
                patch.neighborType[2] = REMOTE;
                patch.neighborPatchIdx[2] = (patch_ny - 1) * patch_nx + px;
                patch.neighborMpiRank[2] = mpi_rank - mpi_nx;
            } else {
                patch.neighborType[2] = LOCAL;
                patch.neighborPatchIdx[2] = p - patch_nx;
                patch.neighborMpiRank[2] = mpi_rank;
            }

            if (patch_ny - 1 == py && mpi_ny - 1 == mpi_y) {
                patch.neighborType[3] = BOUNDARY;
                patch.neighborPatchIdx[3] = -1;
                patch.neighborMpiRank[3] = -1;
            } else if (patch_ny - 1 == py) {
                patch.neighborType[3] = REMOTE;
                patch.neighborPatchIdx[3] = px;
                patch.neighborMpiRank[3] = mpi_rank + mpi_nx;
            } else {
                patch.neighborType[3] = LOCAL;
                patch.neighborPatchIdx[3] = p + patch_nx;
                patch.neighborMpiRank[3] = mpi_rank;
            }

            for (auto i = 0; i < 4; ++i) {
                if (REMOTE == patch.neighborType[i]) {
                    checkCudaError(hipMalloc((void **)&patch.h_d_bufSend[i], sizeof(tpe) * (i < 2 ? ny - 2 : nx - 2)));
                    checkCudaError(hipMalloc((void **)&patch.h_d_bufRecv[i], sizeof(tpe) * (i < 2 ? ny - 2 : nx - 2)));
                } else {
                    patch.h_d_bufSend[i] = nullptr;
                    patch.h_d_bufRecv[i] = nullptr;
                    patch.reqsSend[i] = MPI_REQUEST_NULL;
                    patch.reqsRecv[i] = MPI_REQUEST_NULL;
                }
            }

            checkCudaError(hipMalloc((void **)&patch.d_d_bufSend, sizeof(tpe *) * 4));
            checkCudaError(hipMalloc((void **)&patch.d_d_bufRecv, sizeof(tpe *) * 4));

            checkCudaError(hipMemcpy(patch.d_d_bufSend, patch.h_d_bufSend, sizeof(tpe *) * 4, hipMemcpyHostToDevice));
            checkCudaError(hipMemcpy(patch.d_d_bufRecv, patch.h_d_bufRecv, sizeof(tpe *) * 4, hipMemcpyHostToDevice));
        }
    }

    Patch<tpe> *d_patches;
    checkCudaError(hipMalloc((void **)&d_patches, sizeof(Patch<tpe>) * numPatches));
    checkCudaError(hipMemcpy(d_patches, patches, sizeof(Patch<tpe>) * numPatches, hipMemcpyHostToDevice));

    ncclComm_t *nccl_comms;
    checkCudaError(hipHostMalloc((void **)&nccl_comms, sizeof(ncclComm_t) * numPatches));
    for (auto p = 0; p < numPatches; ++p) {
        ncclUniqueId nccl_uid;
        if (0 == mpi_rank)
            checkNcclError(ncclGetUniqueId(&nccl_uid));
        MPI_Bcast(&nccl_uid, sizeof(ncclUniqueId), MPI_BYTE, 0, MPI_COMM_WORLD);
        MPI_Barrier(MPI_COMM_WORLD);

        checkNcclError(ncclCommInitRank(&nccl_comms[p], mpi_size, nccl_uid, mpi_rank));
    }

    if (0 == mpi_rank) {
        int nccl_version = 0;
        checkNcclError(ncclGetVersion(&nccl_version));
    }

    // init
    initStencil2D(patches, nx, ny, numPatches);

    for (auto p = 0; p < numPatches; ++p) {
        auto& patch = patches[p];
        checkCudaError(hipMemcpy(patch.d_u, patch.u, sizeof(tpe) * nx * ny, hipMemcpyHostToDevice));
        checkCudaError(hipMemcpy(patch.d_uNew, patch.uNew, sizeof(tpe) * nx * ny, hipMemcpyHostToDevice));
    }

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i)
        performIteration(patches, numPatches, nx, ny, mpi_rank, mpi_x, mpi_y, mpi_nx, mpi_ny, nccl_tpe, nccl_comms);
    checkCudaError(hipDeviceSynchronize(), true);

    MPI_Barrier(MPI_COMM_WORLD);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i)
        performIteration(patches, numPatches, nx, ny, mpi_rank, mpi_x, mpi_y, mpi_nx, mpi_ny, nccl_tpe, nccl_comms);
    checkCudaError(hipDeviceSynchronize(), true);

    MPI_Barrier(MPI_COMM_WORLD);

    auto end = std::chrono::steady_clock::now();

    if (0 == mpi_rank)
        printStats<tpe>(end - start, nIt, numPatches * mpi_size * (nx - 2) * (ny - 2), tpeName, sizeof(tpe) + sizeof(tpe), 7);

    for (auto i = 0; i < numPatches; ++i) {
        checkCudaError(hipMemcpy(patches[i].u, patches[i].d_u, sizeof(tpe) * nx * ny, hipMemcpyDeviceToHost));
        checkCudaError(hipMemcpy(patches[i].uNew, patches[i].d_uNew, sizeof(tpe) * nx * ny, hipMemcpyDeviceToHost));
    }

    // check solution
    tpe res = checkSolutionStencil2D(patches, nx, ny, nIt + nItWarmUp, numPatches);
    MPI_Reduce(0 == mpi_rank ? MPI_IN_PLACE : &res, &res, 1, MPI_TPE, MPI_SUM, 0, MPI_COMM_WORLD);
    if (0 == mpi_rank) {
        res = sqrt(res);
        std::cout << "  Final residual is " << res << std::endl;
    }

    for (auto p = 0; p < numPatches; ++p) {
        auto& patch = patches[p];

        for (auto i = 0; i < 4; ++i) {
            if (REMOTE == patch.neighborType[i]) {
                checkCudaError(hipFree(patch.h_d_bufSend[i]));
                checkCudaError(hipFree(patch.h_d_bufRecv[i]));
            }
        }

        checkCudaError(hipFree(patch.d_d_bufSend));
        checkCudaError(hipFree(patch.d_d_bufRecv));

        checkCudaError(hipFree(patch.d_u));
        checkCudaError(hipFree(patch.d_uNew));

        checkCudaError(hipHostFree(patch.u));
        checkCudaError(hipHostFree(patch.uNew));
    }

    checkCudaError(hipFree(d_patches));
    checkCudaError(hipHostFree(patches));

    for (auto p = 0; p < numPatches; ++p)
        checkNcclError(ncclCommDestroy(nccl_comms[p]));
    checkCudaError(hipHostFree(nccl_comms));

    MPI_Finalize();

    return 0;
}


int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cout << "Missing type specification" << std::endl;
        return -1;
    }

    std::string tpeName(argv[1]);

    if ("float" == tpeName)
        return realMain<float>(argc, argv, MPI_FLOAT, ncclFloat);
    if ("double" == tpeName)
        return realMain<double>(argc, argv, MPI_DOUBLE, ncclDouble);

    std::cout << "Invalid type specification (" << argv[1] << "); supported types are" << std::endl;
    std::cout << "  int, long, float, double" << std::endl;
    return -1;
}
