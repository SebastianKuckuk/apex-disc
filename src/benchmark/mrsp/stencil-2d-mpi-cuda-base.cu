#include "hip/hip_runtime.h"
#include "stencil-2d-cuda-util.cuh"
#include "stencil-2d-util.h"

#include <mpi.h>

#include "../../cuda-util.h"


template <typename tpe>
__global__ void stencil2d(const tpe *const __restrict__ u, tpe *__restrict__ uNew, const size_t nx, const size_t ny) {
    const size_t i0 = blockIdx.x * blockDim.x + threadIdx.x + 1;
    const size_t i1 = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (i0 < nx - 1 && i1 < ny - 1) {
        uNew[i0 + i1 * nx] = 0.25 * u[i0 + i1 * nx + 1] + 0.25 * u[i0 + i1 * nx - 1] + 0.25 * u[i0 + nx * (i1 + 1)] + 0.25 * u[i0 + nx * (i1 - 1)];
    }
}

template <typename tpe>
inline void performIteration(tpe *&d_u, tpe *&d_uNew, tpe *h_d_bufSend[4], tpe *h_d_bufRecv[4], const size_t nx, const size_t ny,
                             int mpi_rank, int mpi_x, int mpi_y, int mpi_nx, int mpi_ny, MPI_Datatype MPI_TPE) {

    dim3 blockSize(16, 16);
    dim3 numBlocks(ceilingDivide(nx - 2, blockSize.x), ceilingDivide(ny - 2, blockSize.y));
    int blockSize1D = 32;

    MPI_Request reqs[] = {MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL,
                          MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL};

    if (mpi_x > 0)
        packBufferVertical<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(d_u, h_d_bufSend[0], nx + 1, nx, ny);
    else
        applyBCWest<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(d_u, nx, ny);

    if (mpi_x < mpi_nx - 1)
        packBufferVertical<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(d_u, h_d_bufSend[1], nx + nx - 2, nx, ny);
    else
        applyBCEast<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(d_u, nx, ny);

    if (mpi_y > 0)
        packBufferHorizontal<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(d_u, h_d_bufSend[2], nx + 1, nx, ny);
    else
        applyBCSouth<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(d_u, nx, ny);

    if (mpi_y < mpi_ny - 1)
        packBufferHorizontal<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(d_u, h_d_bufSend[3], (ny - 2) * nx + 1, nx, ny);
    else
        applyBCNorth<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(d_u, nx, ny);

    checkCudaError(hipDeviceSynchronize(), true);

    if (mpi_x > 0) {
        MPI_Isend(h_d_bufSend[0], ny - 2, MPI_TPE, mpi_rank - 1, 0, MPI_COMM_WORLD, &reqs[0]);
        MPI_Irecv(h_d_bufRecv[0], ny - 2, MPI_TPE, mpi_rank - 1, 0, MPI_COMM_WORLD, &reqs[4]);
    }
    if (mpi_x < mpi_nx - 1) {
        MPI_Isend(h_d_bufSend[1], ny - 2, MPI_TPE, mpi_rank + 1, 0, MPI_COMM_WORLD, &reqs[1]);
        MPI_Irecv(h_d_bufRecv[1], ny - 2, MPI_TPE, mpi_rank + 1, 0, MPI_COMM_WORLD, &reqs[5]);
    }
    if (mpi_y > 0) {
        MPI_Isend(h_d_bufSend[2], nx - 2, MPI_TPE, mpi_rank - mpi_nx, 0, MPI_COMM_WORLD, &reqs[2]);
        MPI_Irecv(h_d_bufRecv[2], nx - 2, MPI_TPE, mpi_rank - mpi_nx, 0, MPI_COMM_WORLD, &reqs[6]);
    }
    if (mpi_y < mpi_ny - 1) {
        MPI_Isend(h_d_bufSend[3], nx - 2, MPI_TPE, mpi_rank + mpi_nx, 0, MPI_COMM_WORLD, &reqs[3]);
        MPI_Irecv(h_d_bufRecv[3], nx - 2, MPI_TPE, mpi_rank + mpi_nx, 0, MPI_COMM_WORLD, &reqs[7]);
    }

    if (mpi_x > 0) {
        MPI_Wait(&reqs[4], MPI_STATUS_IGNORE);
        unpackBufferVertical<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(h_d_bufRecv[0], d_u, nx, nx, ny);
    }
    if (mpi_x < mpi_nx - 1) {
        MPI_Wait(&reqs[5], MPI_STATUS_IGNORE);
        unpackBufferVertical<<<ceilingDivide(ny - 2, blockSize1D), blockSize1D>>>(h_d_bufRecv[1], d_u, nx + nx - 1, nx, ny);
    }
    if (mpi_y > 0) {
        MPI_Wait(&reqs[6], MPI_STATUS_IGNORE);
        unpackBufferHorizontal<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(h_d_bufRecv[2], d_u, 1, nx, ny);
    }
    if (mpi_y < mpi_ny - 1) {
        MPI_Wait(&reqs[7], MPI_STATUS_IGNORE);
        unpackBufferHorizontal<<<ceilingDivide(nx - 2, blockSize1D), blockSize1D>>>(h_d_bufRecv[3], d_u, (ny - 1) * nx + 1, nx, ny);
    }

    stencil2d<<<numBlocks, blockSize>>>(d_u, d_uNew, nx, ny);

    MPI_Waitall(4, reqs, MPI_STATUSES_IGNORE);

    std::swap(d_u, d_uNew);
}

template <typename tpe>
inline int realMain(int argc, char *argv[], MPI_Datatype MPI_TPE) {
    MPI_Init(&argc, &argv);

    char *tpeName;
    size_t nx, ny, nItWarmUp, nIt;
    unsigned int mpi_nx, mpi_ny;
    parseCLA_2d(argc, argv, tpeName, nx, ny, nItWarmUp, nIt, mpi_nx, mpi_ny);

    int mpi_rank, mpi_size;
    MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);

    if (mpi_size != mpi_nx * mpi_ny) {
        std::cerr << "Number of MPI processes must be " << mpi_nx * mpi_ny << std::endl;
        MPI_Finalize();
        return -1;
    }

    int mpi_x = mpi_rank % mpi_nx;
    int mpi_y = mpi_rank / mpi_nx;

    int numDevices;
    checkCudaError(hipGetDeviceCount(&numDevices));
    int device = mpi_rank % numDevices;
    checkCudaError(hipSetDevice(device));

    tpe *u;
    checkCudaError(hipHostMalloc((void **)&u, sizeof(tpe) * nx * ny));
    tpe *uNew;
    checkCudaError(hipHostMalloc((void **)&uNew, sizeof(tpe) * nx * ny));

    tpe *d_u;
    checkCudaError(hipMalloc((void **)&d_u, sizeof(tpe) * nx * ny));
    tpe *d_uNew;
    checkCudaError(hipMalloc((void **)&d_uNew, sizeof(tpe) * nx * ny));

    tpe *h_d_bufSend[4];
    tpe *h_d_bufRecv[4];
    for (auto i = 0; i < 4; ++i) {
        checkCudaError(hipMalloc((void **)&h_d_bufSend[i], sizeof(tpe) * (i < 2 ? ny - 2 : nx - 2)));
        checkCudaError(hipMalloc((void **)&h_d_bufRecv[i], sizeof(tpe) * (i < 2 ? ny - 2 : nx - 2)));
    }

    // init
    initStencil2D(u, uNew, nx, ny);

    checkCudaError(hipMemcpy(d_u, u, sizeof(tpe) * nx * ny, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_uNew, uNew, sizeof(tpe) * nx * ny, hipMemcpyHostToDevice));

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i)
        performIteration(d_u, d_uNew, h_d_bufSend, h_d_bufRecv, nx, ny, mpi_rank, mpi_x, mpi_y, mpi_nx, mpi_ny, MPI_TPE);
    checkCudaError(hipDeviceSynchronize(), true);

    MPI_Barrier(MPI_COMM_WORLD);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i)
        performIteration(d_u, d_uNew, h_d_bufSend, h_d_bufRecv, nx, ny, mpi_rank, mpi_x, mpi_y, mpi_nx, mpi_ny, MPI_TPE);
    checkCudaError(hipDeviceSynchronize(), true);

    MPI_Barrier(MPI_COMM_WORLD);

    auto end = std::chrono::steady_clock::now();

    if (0 == mpi_rank)
        printStats<tpe>(end - start, nIt, (nx - 2) * (ny - 2), tpeName, mpi_size * (sizeof(tpe) + sizeof(tpe)), mpi_size * 7);

    checkCudaError(hipMemcpy(u, d_u, sizeof(tpe) * nx * ny, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(uNew, d_uNew, sizeof(tpe) * nx * ny, hipMemcpyDeviceToHost));

    // check solution
    tpe res = checkSolutionStencil2D(u, uNew, nx, ny, nIt + nItWarmUp);
    MPI_Reduce(0 == mpi_rank ? MPI_IN_PLACE : &res, &res, 1, MPI_TPE, MPI_SUM, 0, MPI_COMM_WORLD);
    if (0 == mpi_rank) {
        res = sqrt(res);
        std::cout << "  Final residual is " << res << std::endl;
    }

    for (auto i = 0; i < 4; ++i) {
        checkCudaError(hipFree(h_d_bufSend[i]));
        checkCudaError(hipFree(h_d_bufRecv[i]));
    }

    checkCudaError(hipFree(d_u));
    checkCudaError(hipFree(d_uNew));

    checkCudaError(hipHostFree(u));
    checkCudaError(hipHostFree(uNew));

    MPI_Finalize();

    return 0;
}


int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cout << "Missing type specification" << std::endl;
        return -1;
    }

    std::string tpeName(argv[1]);

    if ("float" == tpeName)
        return realMain<float>(argc, argv, MPI_FLOAT);
    if ("double" == tpeName)
        return realMain<double>(argc, argv, MPI_DOUBLE);

    std::cout << "Invalid type specification (" << argv[1] << "); supported types are" << std::endl;
    std::cout << "  int, long, float, double" << std::endl;
    return -1;
}
