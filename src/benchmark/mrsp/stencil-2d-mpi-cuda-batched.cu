#include "hip/hip_runtime.h"
#include "stencil-2d-cuda-util.cuh"
#include "stencil-2d-util.h"

#include <mpi.h>

#include "../../cuda-util.h"


template <typename tpe>
__global__ void stencil2d(const tpe *const __restrict__ u, tpe *__restrict__ uNew, const size_t nx, const size_t ny) {
    const size_t i0 = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t i1 = blockIdx.y * blockDim.y + threadIdx.y;

    if (i0 >= 1 && i0 < nx - 1 && i1 >= 1 && i1 < ny - 1) {
        uNew[i0 + i1 * nx] = 0.25 * u[i0 + i1 * nx + 1] + 0.25 * u[i0 + i1 * nx - 1] + 0.25 * u[i0 + nx * (i1 + 1)] + 0.25 * u[i0 + nx * (i1 - 1)];
    }
}

template <typename tpe>
__global__ void packAndApplyBC(tpe *__restrict__ u, tpe *__restrict__ sendBuf[4], const size_t nx, const size_t ny,
                               const int mpi_x, const int mpi_y, const int mpi_nx, const int mpi_ny) {
    size_t dir = blockIdx.y * blockDim.y + threadIdx.y;
    size_t idxStart = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idxStride = blockDim.x * gridDim.x;

    auto buf = sendBuf[dir];

    switch (dir) {
    case 0:
        if (mpi_x > 0) {
            for (size_t y = idxStart; y < ny - 2; y += idxStride)
                buf[y] = u[(nx + 1) + y * nx];
        } else {
            for (size_t y = idxStart; y < ny - 2; y += idxStride)
                u[(y + 1) * nx] = 2 * (tpe)0 - u[(y + 1) * nx + 1];
        }
        break;

    case 1:
        if (mpi_x < mpi_nx - 1) {
            for (size_t y = idxStart; y < ny - 2; y += idxStride)
                buf[y] = u[(nx + nx - 2) + y * nx];
        } else {
            for (size_t y = idxStart; y < ny - 2; y += idxStride)
                u[(y + 1) * nx + nx - 1] = 2 * (tpe)0 - u[(y + 1) * nx + nx - 2];
        }
        break;

    case 2:
        if (mpi_y > 0) {
            for (size_t x = idxStart; x < nx - 2; x += idxStride)
                buf[x] = u[(nx + 1) + x];
        } else {
            for (size_t x = idxStart; x < nx - 2; x += idxStride)
                u[x + 1] = 2 * (tpe)0 - u[nx + x + 1];
        }
        break;

    case 3:
        if (mpi_y < mpi_ny - 1) {
            for (size_t x = idxStart; x < nx - 2; x += idxStride)
                buf[x] = u[(ny - 2) * nx + 1 + x];
        } else {
            for (size_t x = idxStart; x < nx - 2; x += idxStride)
                u[(ny - 1) * nx + x + 1] = 2 * (tpe)0 - u[(ny - 2) * nx + x + 1];
        }
        break;
    }
}

template <typename tpe>
__global__ void unpack(tpe *__restrict__ u, tpe *__restrict__ recvBuf[4], const size_t nx, const size_t ny,
                       const int mpi_x, const int mpi_y, const int mpi_nx, const int mpi_ny) {
    size_t dir = blockIdx.y * blockDim.y + threadIdx.y;
    size_t idxStart = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idxStride = blockDim.x * gridDim.x;

    auto buf = recvBuf[dir];

    switch (dir) {
    case 0:
        if (mpi_x > 0)
            for (size_t y = idxStart; y < ny - 2; y += idxStride)
                u[nx + y * nx] = buf[y];
        break;

    case 1:
        if (mpi_x < mpi_nx - 1)
            for (size_t y = idxStart; y < ny - 2; y += idxStride)
                u[nx + nx - 1 + y * nx] = buf[y];
        break;

    case 2:
        if (mpi_y > 0)
            for (size_t x = idxStart; x < nx - 2; x += idxStride)
                u[1 + x] = buf[x];
        break;

    case 3:
        if (mpi_y < mpi_ny - 1)
            for (size_t x = idxStart; x < nx - 2; x += idxStride)
                u[(ny - 1) * nx + 1 + x] = buf[x];
        break;
    }
}

template <typename tpe>
inline void performIteration(tpe *&d_u, tpe *&d_uNew, tpe *d_d_bufSend[4], tpe *d_d_bufRecv[4], tpe *h_d_bufSend[4], tpe *h_d_bufRecv[4],
                             const size_t nx, const size_t ny,
                             int mpi_rank, int mpi_x, int mpi_y, int mpi_nx, int mpi_ny, MPI_Datatype MPI_TPE) {

    dim3 blockSize(16, 16);
    dim3 numBlocks(ceilingDivide(nx, blockSize.x), ceilingDivide(ny, blockSize.y));
    int blockSize1D = 32;

    MPI_Request reqs[] = {MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL,
                          MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL, MPI_REQUEST_NULL};

    packAndApplyBC<<<dim3(ceilingDivide(max(nx, ny) - 2, blockSize1D), 4), dim3(blockSize1D, 1)>>>(d_u, d_d_bufSend, nx, ny, mpi_x, mpi_y, mpi_nx, mpi_ny);

    checkCudaError(hipDeviceSynchronize(), true);

    if (mpi_x > 0) {
        MPI_Isend(h_d_bufSend[0], ny - 2, MPI_TPE, mpi_rank - 1, 0, MPI_COMM_WORLD, &reqs[0]);
        MPI_Irecv(h_d_bufRecv[0], ny - 2, MPI_TPE, mpi_rank - 1, 0, MPI_COMM_WORLD, &reqs[4]);
    }
    if (mpi_x < mpi_nx - 1) {
        MPI_Isend(h_d_bufSend[1], ny - 2, MPI_TPE, mpi_rank + 1, 0, MPI_COMM_WORLD, &reqs[1]);
        MPI_Irecv(h_d_bufRecv[1], ny - 2, MPI_TPE, mpi_rank + 1, 0, MPI_COMM_WORLD, &reqs[5]);
    }
    if (mpi_y > 0) {
        MPI_Isend(h_d_bufSend[2], nx - 2, MPI_TPE, mpi_rank - mpi_nx, 0, MPI_COMM_WORLD, &reqs[2]);
        MPI_Irecv(h_d_bufRecv[2], nx - 2, MPI_TPE, mpi_rank - mpi_nx, 0, MPI_COMM_WORLD, &reqs[6]);
    }
    if (mpi_y < mpi_ny - 1) {
        MPI_Isend(h_d_bufSend[3], nx - 2, MPI_TPE, mpi_rank + mpi_nx, 0, MPI_COMM_WORLD, &reqs[3]);
        MPI_Irecv(h_d_bufRecv[3], nx - 2, MPI_TPE, mpi_rank + mpi_nx, 0, MPI_COMM_WORLD, &reqs[7]);
    }

    MPI_Waitall(4, &reqs[4], MPI_STATUSES_IGNORE);
    unpack<<<dim3(ceilingDivide(max(nx, ny) - 2, blockSize1D), 4), dim3(blockSize1D, 1)>>>(d_u, d_d_bufRecv, nx, ny, mpi_x, mpi_y, mpi_nx, mpi_ny);

    stencil2d<<<numBlocks, blockSize>>>(d_u, d_uNew, nx, ny);

    MPI_Waitall(4, reqs, MPI_STATUSES_IGNORE);

    std::swap(d_u, d_uNew);
}

template <typename tpe>
inline int realMain(int argc, char *argv[], MPI_Datatype MPI_TPE) {
    MPI_Init(&argc, &argv);

    char *tpeName;
    size_t nx, ny, nItWarmUp, nIt;
    unsigned int mpi_nx, mpi_ny;
    parseCLA_2d(argc, argv, tpeName, nx, ny, nItWarmUp, nIt, mpi_nx, mpi_ny);

    int mpi_rank, mpi_size;
    MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);

    if (mpi_size != mpi_nx * mpi_ny) {
        std::cerr << "Number of MPI processes must be " << mpi_nx * mpi_ny << std::endl;
        MPI_Finalize();
        return -1;
    }

    int mpi_x = mpi_rank % mpi_nx;
    int mpi_y = mpi_rank / mpi_nx;

    int numDevices;
    checkCudaError(hipGetDeviceCount(&numDevices));
    int device = mpi_rank % numDevices;
    checkCudaError(hipSetDevice(device));

    tpe *u;
    checkCudaError(hipHostMalloc((void **)&u, sizeof(tpe) * nx * ny));
    tpe *uNew;
    checkCudaError(hipHostMalloc((void **)&uNew, sizeof(tpe) * nx * ny));

    tpe *d_u;
    checkCudaError(hipMalloc((void **)&d_u, sizeof(tpe) * nx * ny));
    tpe *d_uNew;
    checkCudaError(hipMalloc((void **)&d_uNew, sizeof(tpe) * nx * ny));

    tpe *h_d_bufSend[4];
    tpe *h_d_bufRecv[4];
    for (auto i = 0; i < 4; ++i) {
        checkCudaError(hipMalloc((void **)&h_d_bufSend[i], sizeof(tpe) * (i < 2 ? ny - 2 : nx - 2)));
        checkCudaError(hipMalloc((void **)&h_d_bufRecv[i], sizeof(tpe) * (i < 2 ? ny - 2 : nx - 2)));
    }

    tpe **d_d_bufSend;
    tpe **d_d_bufRecv;
    checkCudaError(hipMalloc((void **)&d_d_bufSend, sizeof(tpe *) * 4));
    checkCudaError(hipMalloc((void **)&d_d_bufRecv, sizeof(tpe *) * 4));
    checkCudaError(hipMemcpy(d_d_bufSend, h_d_bufSend, sizeof(tpe *) * 4, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_d_bufRecv, h_d_bufRecv, sizeof(tpe *) * 4, hipMemcpyHostToDevice));

    // init
    initStencil2D(u, uNew, nx, ny);

    checkCudaError(hipMemcpy(d_u, u, sizeof(tpe) * nx * ny, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_uNew, uNew, sizeof(tpe) * nx * ny, hipMemcpyHostToDevice));

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i)
        performIteration(d_u, d_uNew, d_d_bufSend, d_d_bufRecv, h_d_bufSend, h_d_bufRecv, nx, ny, mpi_rank, mpi_x, mpi_y, mpi_nx, mpi_ny, MPI_TPE);
    checkCudaError(hipDeviceSynchronize(), true);

    MPI_Barrier(MPI_COMM_WORLD);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt + 1; ++i)
        performIteration(d_u, d_uNew, d_d_bufSend, d_d_bufRecv, h_d_bufSend, h_d_bufRecv, nx, ny, mpi_rank, mpi_x, mpi_y, mpi_nx, mpi_ny, MPI_TPE);
    checkCudaError(hipDeviceSynchronize(), true);

    MPI_Barrier(MPI_COMM_WORLD);

    auto end = std::chrono::steady_clock::now();

    if (0 == mpi_rank)
        printStats<tpe>(end - start, nIt, nx * ny, tpeName, mpi_size * (sizeof(tpe) + sizeof(tpe)), mpi_size * 7);

    checkCudaError(hipMemcpy(u, d_u, sizeof(tpe) * nx * ny, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(uNew, d_uNew, sizeof(tpe) * nx * ny, hipMemcpyDeviceToHost));

    // check solution
    tpe res = checkSolutionStencil2D(u, uNew, nx, ny, nIt + nItWarmUp);
    MPI_Reduce(0 == mpi_rank ? MPI_IN_PLACE : &res, &res, 1, MPI_TPE, MPI_SUM, 0, MPI_COMM_WORLD);
    if (0 == mpi_rank) {
        res = sqrt(res);
        std::cout << "  Final residual is " << res << std::endl;
    }

    for (auto i = 0; i < 4; ++i) {
        checkCudaError(hipFree(h_d_bufSend[i]));
        checkCudaError(hipFree(h_d_bufRecv[i]));
    }
    checkCudaError(hipFree(d_d_bufSend));
    checkCudaError(hipFree(d_d_bufRecv));

    checkCudaError(hipFree(d_u));
    checkCudaError(hipFree(d_uNew));

    checkCudaError(hipHostFree(u));
    checkCudaError(hipHostFree(uNew));

    MPI_Finalize();

    return 0;
}


int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cout << "Missing type specification" << std::endl;
        return -1;
    }

    std::string tpeName(argv[1]);

    if ("float" == tpeName)
        return realMain<float>(argc, argv, MPI_FLOAT);
    if ("double" == tpeName)
        return realMain<double>(argc, argv, MPI_DOUBLE);

    std::cout << "Invalid type specification (" << argv[1] << "); supported types are" << std::endl;
    std::cout << "  int, long, float, double" << std::endl;
    return -1;
}
